#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
using namespace std;


__global__ void multKernel(int *a, int *b, int *c, int n)
{
    //Calculating rows and columns for a particular thread
    int row = (blockIdx.x * blockDim.y) + threadIdx.x; //Calculating row
    int col = (blockIdx.x * blockDim.x) + threadIdx.x; //Calculating column

    int sum = 0;

    //Checking boundary condition
    if ((row < n) && (col < n)) {
        for (int k = 0; k < n; k++) {
            sum += a[row * n + k] * b[k * n + col];
        }
        c[row * n + col] = sum;
    }
}

int main()
{
    //Matrix of size 1024 x 1024
    //int n = 1 << 10;
    constexpr int n = 1 << 10; 

    //Size (in bytes) of mathrix
    //size_t bytes = n * n * sizeof(int); 
    constexpr size_t bytes = n * n * sizeof(int); 

    //CPU pointers
    int* a, * b, * c; 

    // Allocating memory for these host pointers
    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);
    hipMallocManaged(&c, bytes);

    /*
    //Allocate CPU memory
    a = (int*)malloc(bytes);
    b = (int*)malloc(bytes);
    c = (int*)malloc(bytes);*/

    // device ID (of GPU) for prefetching
    int id = hipGetDevice(&id);

    /*
    //Device pointer
    int* da, * db, * dc;
    
    //Allocate device memory
    hipMalloc(&da, bytes);
    hipMalloc(&db, bytes);
    hipMalloc(&dc, bytes);
    */

    //Generating random matrix
    for(int i =0; i < n; i++)
        for (int j = 0; j < n; j++) {
            a[i * n + j] = rand() % 100;
            b[i * n + j] = rand() % 100;
        }
    /*
    //Copy data to device from host
    hipMemcpy(da, a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(db, b, bytes, hipMemcpyHostToDevice);
    */

    //Thread per blocks
    int block_size = 16; //Since we are using 2d array, therefore 16 * 16 = 256 threads per block

    //Blocks in each dimensions
    int grid_size = (int)ceil(n / block_size); // Dividing total elements by number of threads to get blocks for each element

    dim3 grid(grid_size, grid_size); //Dimension of grid
    dim3 threads(block_size, block_size); //Dimension of block

    // prefetching 'a', 'b' and 'c'
    hipMemPrefetchAsync(a, bytes, id);
    hipMemPrefetchAsync(b, bytes, id);
    hipMemPrefetchAsync(c, bytes, id);

    //Launching kernel
    multKernel <<< grid, threads >>> (a, b, c, n);

    //Synchronization needed because to make sure kernel is done
    hipDeviceSynchronize();

    /*
    //Copy back to host
    hipMemcpy(c, dc, bytes, hipMemcpyDeviceToHost);
    */

    //Prefetching back to CPU because we know that kernel is completed
    //Needed when we are comparing the matrix between GPU and CPU
    hipMemPrefetchAsync(c, bytes, hipCpuDeviceId); //hipCpuDeviceId we don't need to calculate, the system automatically knows it

    cout << "Done successfully" << endl;
    
    //Free up the memory
    hipFree(a);
    hipFree(b);
    hipFree(c);

    return 0;
}
