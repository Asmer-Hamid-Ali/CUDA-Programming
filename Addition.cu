#include "hip/hip_runtime.h"


#include <stdio.h>
#include <malloc.h>


__global__ void add(int *x, int *y, int *z)
{
    *z = *x + *y;
    printf("z is %d\n", *z);
}


int main()
{
    //Declaration
    int *a, *b, *c;
    int *deva, *devb, *devc;

    //Dynamic Memory Allocation in Host
    a = (int *)malloc(sizeof(int));
    b = (int *)malloc(sizeof(int));
    c = (int *)malloc(sizeof(int));

    //Reserving Memory in Device
    hipMalloc((int **)&deva, sizeof(int));
    hipMalloc((int **)&devb, sizeof(int));
    hipMalloc((int **)&devc, sizeof(int));


    //Inputting values from user
    printf("Enter value of a and b\n");
    scanf("%d %d", a, b);

    /**c = *a + *b;
    printf("answer: %d\n", *c);*/

    //Coping values from HostToDevice
    hipMemcpy(deva, a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devb, b, sizeof(int), hipMemcpyHostToDevice);

    //Calling Kernel
    add<<<1,1>>>(deva, devb, devc);

    //Coping values from DeviceToHost
    hipMemcpy(c, devc, sizeof(int), hipMemcpyDeviceToHost);

    printf("Result is: %d\n", *c);

    //Free-up the memory
    hipFree(deva), hipFree(devb), hipFree(devc);

    return 0;
}
